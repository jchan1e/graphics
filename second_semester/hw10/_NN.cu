#include "hip/hip_runtime.h"
#include "reader_.c"
#include <math.h>
#include <time.h>


float sigmoid(float x)
{
   return 1/(1+exp(-x));
}

float dsigmoid(float x)
{
   return x*(1-x);
}

void err(hipError_t returnVal)
{
   if (returnVal != hipSuccess)
   {
      fprintf(stderr, "CUDA Failure: %s\n", hipGetErrorString(returnVal));
      exit(EXIT_FAILURE);
   }
}

__global__ void train(const float* in, const float* label, const float* syn1, const float* syn2, float* dsyn1, float* dsyn2, const float alpha)
{
   int i = blockDim.x*blockIdx.x + threadIdx.x;

   if (i < 60000)
   {
      float layer1[100] = {0.0};            //Middle layer
      float outs[10] = {0.0};               //Output layer

      // Forward pass
      //input to middle layer
#pragma unroll 28
      for (int j=0; j < 28; ++j)
      {
#pragma unroll 28
         for (int k=0; k < 28; ++k)
         {
#pragma unroll 100
            for (int l=0; l < 100; ++l)
            {
               layer1[l] += in[i*28*28 + j*28 + k] * syn1[j*28*100 + k*100 + l];
            }
         }
      }
#pragma unroll 100
      for (int j=0; j < 100; ++j)
         layer1[j] = 1/(1+exp(-layer1[j]));
      
      //middle to output layer
#pragma unroll 100
      for (int j=0; j < 100; ++j)
      {
#pragma unroll 10
         for (int k=0; k < 10; ++k)
         {
            outs[k] += layer1[j] * syn2[j*10 + k];
         }
      }
#pragma unroll 10
      for (int j=0; j < 10; ++j)
      {
         outs[j] = 1/(1+exp(-outs[j]));
      }

      //Back Propagation
      //   error[k] = labels[i][k] - outs[k]
      //   delta[k] = error * dsigmoid(outs[k])
      //   weights2[j][k] += delta[k] * layer1[j]
      //output to middle
      float delta2[10] = {0};
      //__shared__ float psyn2[10*100];
#pragma unroll 100
      for (int j=0; j < 100; ++j)
      {
#pragma unroll 10
         for (int k=0; k < 10; ++k)
         {
            delta2[k] = (label[i*10 + k] - outs[k]) * outs[k]*(1.0-outs[k]);
            atomicAdd(&dsyn2[j*10 + k], delta2[k] * alpha * layer1[j] / (gridDim.x*blockDim.x));
         }
      }
      //middle to input
      float delta1[100] = {0.0};
      float error1[100] = {0.0};
      //__shared__ float psyn1[28*28*100];
#pragma unroll 28
      for (int h=0; h < 28; ++h)
      {
#pragma unroll 28
         for (int j=0; j < 28; ++j)
         {
#pragma unroll 100
            for (int k=0; k < 100; ++k)
            {
               error1[k] = 0.0;
#pragma unroll 10
               for (int l=0; l < 10; ++l)
                  error1[k] += delta2[l] * syn2[k*10 * l];
               delta1[k] = error1[k] * layer1[k]*(1.0-layer1[k]);
               atomicAdd(&dsyn1[h*28*100 + j*100 + k], alpha * (delta1[k] * in[i*28*28 + h*28 + j] / (gridDim.x*blockDim.x)));
            }
         }
      }
      //__syncthreads();

      //if (threadIdx.x == 0)
      //{
      //   for (int j=0; j < 100; ++j)
      //   {
      //      for (int k=0; k < 10; ++k)
      //      {
      //         atomicAdd(&dsyn2[j*10 + k], psyn2[j*10 + k]);
      //      }
      //   }
      //   for (int h=0; h < 28; ++h)
      //   {
      //      for (int j=0; j < 28; ++j)
      //      {
      //         for (int k=0; k < 100; ++k)
      //         {
      //            atomicAdd(&dsyn1[h*28*100 + j*100 + k], psyn1[h*28*100 + j*100 + k]);
      //         }
      //      }
      //   }
      //}
   }
}

//__global__ void apply(float* out, float* in, int n)
//{
//   int i = blockDim.x*blockIdx.x + threadIdx.x;
//   if (i < n)
//   {
//      atomicAdd(&out[i], in[i]);
//      in[i] = 0.0;
//   }
//}

int main(int argc, char** argv)
{
   if (argc != 7)
   {
      printf("usage: run trainingImages trainingLabels testImages testLabels iterations alpha\n");
      return 2;
   }
   struct data Data = read(argv[1], argv[2]);      //training data
   struct data Test = read(argv[3], argv[4]);      //test data

   float weights1[28*28*100];   //input to middle layer weights
   //float dweights1[28*28*100];  //input to middle layer weights
   float layer1[100];                              //Middle layer
   float weights2[100*10];                        //middle to output layer weights
   //float dweights2[100*10];                       //middle to output layer weights
   float outs[10];                                 //Output layer
   float alpha = atof(argv[6]);

   float* d_in;    err(hipMalloc((void**)&d_in,    28*28*60000*sizeof(float)));
   float* d_label; err(hipMalloc((void**)&d_label, 60000*10*sizeof(float)));
   float* d_syn1;  err(hipMalloc((void**)&d_syn1,  28*28*100*sizeof(float)));
   float* d_dsyn1; err(hipMalloc((void**)&d_dsyn1, 28*28*100*sizeof(float)));
   float* d_syn2;  err(hipMalloc((void**)&d_syn2,  100*10*sizeof(float)));
   float* d_dsyn2; err(hipMalloc((void**)&d_dsyn2, 100*10*sizeof(float)));

   //Initialize weights to random values
   //printf("randomizing initial weights\n");
   srand(1); //make the random values the same each time
   for (int i=0; i < 28; ++i)
   {
      for (int j=0; j < 28; ++j)
      {
         for (int k=0; k < 100; ++k)
         {
            weights1[i*28*100 + j*100 + k] = (float)rand()/(RAND_MAX/2.0) - 1.0;
            //dweights1[i*28*100 + j*100 + k] = 0.0;
         }
      }
   }
   for (int i=0; i<100; ++i)
   {
      for (int j=0; j < 10; ++j)
      {
         weights2[i*10 + j] = (float)rand()/(RAND_MAX/2.0) - 1.0;
         //dweights2[i*10 + j] = 0.0;
      }
   }

   clock_t start = clock();

   err(hipMemcpy(d_in, Data.Image, 28*28*60000*sizeof(float), hipMemcpyHostToDevice));
   err(hipMemcpy(d_label, Data.Label, 10*60000*sizeof(float), hipMemcpyHostToDevice));
   err(hipMemcpy(d_syn1, weights1, 28*28*100*sizeof(float), hipMemcpyHostToDevice));
   err(hipMemcpy(d_dsyn1, d_syn1, 28*28*100*sizeof(float), hipMemcpyDeviceToDevice));
   err(hipMemcpy(d_syn2, weights2, 10*100*sizeof(float), hipMemcpyHostToDevice));
   err(hipMemcpy(d_dsyn2, d_syn2, 10*100*sizeof(float), hipMemcpyDeviceToDevice));

   //float* testI = (float*)malloc(28*28*60000*sizeof(float));
   //float* testL = (float*)malloc(10*60000*sizeof(float));
   //bool potato = false;
   //err(hipMemcpy(testI, d_in, 28*28*60000*sizeof(float), hipMemcpyDeviceToHost));
   //err(hipMemcpy(testL, d_label, 10*60000*sizeof(float), hipMemcpyDeviceToHost));
   //for (int i=0; i < 28*28*60000; ++i)
   //{
   //   if (testI[i] > 1.0 || testI[i] < 0.0)
   //      potato = true;
   //}
   //if (potato)
   //   printf("error in the input data\n");
   //potato = false;
   //for (int i=0; i < 10*60000; ++i)
   //{
   //   if (testL[i] > 1.0 || testL[i] < 0.0)
   //      potato = true;
   //}
   //if (potato)
   //   printf("error in the label data\n");
   //potato = false;

   //train
   //printf("training\n");
   int iterations = atoi(argv[5]);
   float al = 1.0;
   for (int iter=0; iter<iterations; ++iter)
   {
      if (iter < 8)
         al = alpha*20.0;
      else if (iter < 16)
         al = alpha*10.0;
      else if (iter < 32)
         al = alpha*5.0;
      else if (iter < 64)
         al = alpha*2.0;
      else
         al = alpha;
      train<<<48,   125>>>(&d_in[6000*(iter%10)], &d_label[6000*(iter%10)], d_syn1, d_syn2, d_dsyn1, d_dsyn2, al);
      err(hipMemcpy(d_syn1, d_dsyn1, sizeof(float)*28*28*100, hipMemcpyDeviceToDevice));
      err(hipMemcpy(d_syn2, d_dsyn2, sizeof(float)*100*10,    hipMemcpyDeviceToDevice));

      //err(hipMemcpy(testI, d_in, 28*28*60000*sizeof(float), hipMemcpyDeviceToHost));
      //err(hipMemcpy(testL, d_label, 10*60000*sizeof(float), hipMemcpyDeviceToHost));
      //err(hipMemcpy(weights1, d_syn1, 28*28*100*sizeof(float), hipMemcpyDeviceToHost));
      //err(hipMemcpy(weights2, d_syn2, 10*100*sizeof(float), hipMemcpyDeviceToHost));
      //for (int i=0; i < 28*28*60000; ++i)
      //{
      //   if (testI[i] > 1.0 || testI[i] < 0.0 || isnan(testI[i]))
      //      potato = true;
      //}
      //if (potato)
      //   printf("error in the input data\n");
      //potato = false;
      //for (int i=0; i < 10*60000; ++i)
      //{
      //   if (testL[i] > 1.0 || testL[i] < 0.0 || isnan(testL[i]))
      //      potato = true;
      //}
      //if (potato)
      //   printf("error in the label data\n");
      //potato = false;
      //for (int i=0; i < 28*28*100; ++i)
      //{
      //   if (isnan(weights1[i]))
      //   {
      //      potato = true;
      //      printf("syn1: %d: %f\n", i, weights1[i]);
      //   }
      //}
      //potato = false;
      //for (int i=0; i < 10*100; ++i)
      //{
      //   if (isnan(weights2[i]))
      //   {
      //      potato = true;
      //      printf("syn1: %d: %f\n", i, weights2[i]);
      //   }
      //}
   }
   //free(testI);
   //free(testL);
   err(hipMemcpy(weights1, d_syn1, sizeof(float)*28*28*100, hipMemcpyDeviceToHost));
   err(hipMemcpy(weights2, d_syn2, sizeof(float)*100*10,    hipMemcpyDeviceToHost));

   clock_t diff = clock() - start;
   diff = diff*1000/CLOCKS_PER_SEC;
   printf("computation time: %ld.%ld\n", diff/1000, diff%1000);

   //test
   //printf("testing\n");
   float error = 0.0;
   //for (int i=0; i < 100; ++i)
   //{
   //   for (int j=0; j < 10; ++j)
   //   {
   //      printf("%f ", weights2[i*10 + j]);
   //   }
   //   printf("\n");
   //}
   for (int i=0; i < Test.count; ++i)
   {

      //reset layer states
      for (int j=0; j < 100; ++j)
         layer1[j] = 0.0;
      for (int j=0; j < 10; ++j)
         outs[j] = 0.0;

      // Forward pass
      //input to middle layer
      for (int j=0; j < Test.height; ++j)
      {
         for (int k=0; k < Test.width; ++k)
         {
            for (int l=0; l < 100; ++l)
            {
               layer1[l] += Test.Image[i*28*28 + j*28 + k] * weights1[j*28*100 + k*100 + l];
            }
         }
      }
      for (int j=0; j < 100; ++j)
         layer1[j] = sigmoid(layer1[j]);

      //middle to output layer
      for (int j=0; j < 100; ++j)
      {
         for (int k=0; k < 10; ++k)
         {
            outs[k] += layer1[j] * weights2[j*10 + k];
         }
      }
      for (int j=0; j < 10; ++j)
      {
         outs[j] = sigmoid(outs[j]);
         //printf("%f ", outs[j]);
      }
      //printf("\n");

      //sum up error
      for (int j=0; j < 10; ++j)
      {
         //printf("%f ", Test.Label[i*10 + j]);
         error += fabs(Test.Label[i*10 + j] - outs[j])/10.0;
      }
      //printf("\n");
   }
   //printf("Error: %f\n", error);
   error /= Test.count;
   printf("Error: %f %%\n", error*100.0);

   //clean up data arrays
   //for (int i=0; i<60000; ++i)
   //{
   //   for (int j=0; j<28; ++j)
   //   {
   //      free(Data.Image[i][j]);
   //   }
   //   free(Data.Image[i]);
   //   free(Data.Label[i]);
   //}
   free(Data.Image);
   free(Data.Label);
   //for (int i=0; i<Test.count; ++i)
   //{
   //   for (int j=0; j<Test.height; ++j)
   //   {
   //      free(Test.Image[i][j]);
   //   }
   //   free(Test.Image[i]);
   //   free(Test.Label[i]);
   //}
   free(Test.Image);
   free(Test.Label);

   err(hipFree(d_in));
   err(hipFree(d_label));
   err(hipFree(d_syn1));
   err(hipFree(d_syn2));
   err(hipFree(d_dsyn1));
   err(hipFree(d_dsyn2));

   return EXIT_SUCCESS;
}
